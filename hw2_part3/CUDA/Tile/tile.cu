#include "hip/hip_runtime.h"
/*
    CS205 HW2 Part 3.1
    The following code was written with reference to nVidia tutorial
    and Programming Massively Parallel Processors.
*/

#include <stdio.h>
#include <hipblas.h>

/* define computation parameters */
#define TILE_WIDTH 32 /* tile width for matrix multiplication */
#define MAT_SIZE 1024 /* matrix size in 1 dimension assuming square */
#define BLOCK_SIZE_X 32 /* number of threads in a block in x direction */
#define BLOCK_SIZE_Y 32 /* number of threads in a block in y direction */

/* GPU kernel: tile matrix multiplcation */
__global__ void tile_matmul(float const * const mat_a, float const * const mat_b, float * const mat_c, const int mat_size){

    /* save indices into registers */
    int tIdx_x = threadIdx.x; /* thread index in x */
    int tIdx_y = threadIdx.y; /* thread index in y */
    int bIdx_x = blockIdx.x; /* block index in x */
    int bIdx_y = blockIdx.y; /*block index in y */
    
    /* row and column of mat_c to calculate now */
    int idx_row = bIdx_y * TILE_WIDTH + tIdx_y;
    int idx_col = bIdx_x * TILE_WIDTH + tIdx_x;
    
    /* store tiles of input mat_a and mat_b in shared memory */
    __shared__ float shared_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float shared_b[TILE_WIDTH][TILE_WIDTH];
    
    /* store temp sum */
    float temp_sum = 0;
    
    /* loop through all tile phases */
    for (int tile_phase = 0; tile_phase < mat_size/TILE_WIDTH; tile_phase++){
        
        /* load a tile of mat_a and mat_b from global memory to shared memory */
        shared_a[tIdx_y][tIdx_x] = mat_a[idx_row * mat_size + tile_phase * TILE_WIDTH + tIdx_x];
        shared_b[tIdx_y][tIdx_x] = mat_b[(tile_phase * TILE_WIDTH + tIdx_y) * mat_size + idx_col];
        /* synchronize threads in a tile */
        __syncthreads();
        
        for (int idx_el = 0; idx_el < TILE_WIDTH; idx_el++){
            temp_sum += shared_a[tIdx_y][idx_el] * shared_b[idx_el][tIdx_x];
        }
        __syncthreads();
    }
    mat_c[idx_row * mat_size + idx_col] = temp_sum;
}


/* function to randomly assign values for matrix */
void randmat(float * matrix, int nsize){
    for(int idx=0; idx < nsize * nsize; idx++){
    matrix[idx] = double(rand())/ (double(RAND_MAX) + 1.0);
    }
}


/* function to calculate max error between mat_1 and mat_2 */
void max_err(float * mat_1, float * mat_2, int nsize){
    float err = 0.0;
    for(int idx=0; idx < nsize * nsize; idx++){
        err = max(err, abs(( (float)mat_1[idx] - (float)mat_2[idx]) / (float)mat_1[idx] ) );
    }
    printf("Max error is %e percent \n", err*100.0);
}

/* main function */
int main(int argc, char *argv[]){

    /* check GPU info on Odyssey */
    int dev;
    hipDeviceProp_t prop;

    hipGetDevice(&dev);
    hipGetDeviceProperties(&prop, dev);
    printf(" --- General info --- \n");
    printf("GPU name: %d %s\n", dev, prop.name);
    printf("Compute capability:  %d.%d\n", prop.major, prop.minor);
    printf("Clock rate:  %d\n", prop.clockRate);

    printf(" --- Memory info --- \n");
    printf("Total global mem:  %ld\n", prop.totalGlobalMem);

    printf(" --- Multiprocessor (MP) Info --- \n");
    printf("MP count: %d\n", prop.multiProcessorCount);
    printf("Shared mem per MP:  %ld\n", prop.sharedMemPerBlock);
    printf("Registers per MP:  %d\n", prop.regsPerBlock);
    printf("Threads in warp:  %d\n", prop.warpSize);
    printf("Max threads per block:  %d\n", prop.maxThreadsPerBlock);
    printf("Max thread dimensions:  (%d, %d, %d)\n", prop.maxThreadsDim[0],
        prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0],
        prop.maxGridSize[1], prop.maxGridSize[2]);
    printf( "\n" );

    /* print matrix size n_now */
    const int n_now = MAT_SIZE;
    printf(" --- Matrix multiplicatoin --- \n");
    printf("Matrix size n is %d\n", n_now);

    /* define matrix in host and in device */
    float *host_a, *host_b, *c_cublas, *c_naive;
    float *dev_a, *dev_b, *dev_c;

    /* size of matrix in byte */
    size_t mat_in_byte = (size_t)n_now * (size_t)n_now * sizeof(float);

    /* randomly setup host_a and host_b */
    host_a = (float *) malloc(mat_in_byte);
    host_b = (float *) malloc(mat_in_byte);
    randmat(host_a, n_now);
    randmat(host_b, n_now);

    /* set c_cublas and c_naive to zeros */
    c_cublas = (float *) malloc(mat_in_byte);
    c_naive = (float *) malloc(mat_in_byte);
    memset(c_cublas, 0, mat_in_byte);
    memset(c_naive, 0, mat_in_byte);

    /* setup dev_a, dev_b, and dev_c */
    hipMalloc((void **)&dev_a, mat_in_byte);
    hipMalloc((void **)&dev_b, mat_in_byte);
    hipMalloc((void **)&dev_c, mat_in_byte);
    hipMemcpy(dev_a, host_a, mat_in_byte, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, host_b, mat_in_byte, hipMemcpyHostToDevice);
    hipMemset(dev_c, 0, mat_in_byte);

    /* setup CUDA timer */
    float time_tile, time_cublas;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /*
        tile_matmul kernel computation
    */

    /* preparation */
    dim3 dim_thread(BLOCK_SIZE_X, BLOCK_SIZE_Y, 1);
    dim3 dim_block( ceil(n_now / BLOCK_SIZE_X), ceil(n_now / BLOCK_SIZE_Y), 1);

    /* start timer */
    hipEventRecord(start, 0);

    /* core computation */
    tile_matmul<<<dim_block, dim_thread>>> (dev_a, dev_b, dev_c, n_now);
    
    /* end timer */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_tile, start, stop);

    /* wrap up */
    hipMemcpy(c_naive, dev_c, mat_in_byte, hipMemcpyDeviceToHost);
    hipMemset(dev_c, 0, mat_in_byte);
    printf("tile_matmul elapsed time is %f seconds\n", time_tile / 1000.0f);
    printf("Throughput is %f GFlop/s\n", 2.0 * (double)n_now * (double)n_now * (double)n_now /
    ((double)time_tile * 1.e-3) * 1.e-9);


    /*
        CUBLAS computation
    */

    /* preparation */
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0;
    float beta = 0.0;

    /* start timer */
    hipEventRecord(start, 0);

    /* core computation */
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n_now, n_now, n_now,
        (float *)&alpha, (float *)dev_b, n_now, (float *)dev_a, n_now,
        (float *)&beta, (float *)dev_c, n_now);
    hipDeviceSynchronize();

    /* end timer */
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_cublas, start, stop);

    /* wrap up */
    hipMemcpy(c_cublas, dev_c, mat_in_byte, hipMemcpyDeviceToHost);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipblasDestroy(handle);
    printf("CUBLAS elapsed time is %f seconds\n", time_cublas / 1000.0f);
    printf("Throughput is %f GFlop/s\n", 2.0 * (double)n_now * (double)n_now * (double)n_now /
    ((double)time_cublas * 1.e-3) * 1.e-9);

    /* compare computation results */
    max_err(c_cublas, c_naive, n_now);

    /* clean up memory */
    free(host_a);
    free(host_b);
    free(c_cublas);
    free(c_naive);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipDeviceReset();

    return 0;
}
