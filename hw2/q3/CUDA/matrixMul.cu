#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <string.h>

#define BLOCK_SIZE 16

void printDeviceProp(const hipDeviceProp_t &prop) {
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %lu.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %lu.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %lu.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %lu.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %lu.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

bool InitCUDA() {
    int count;
    hipGetDeviceCount(&count);
    if (count == 0)  {
        fprintf(stderr, "There is no device.\n");
        return false;
    }
    int i;
    for (i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printDeviceProp(prop);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if (prop.major >= 1) {
                break;
            }
        }
    }
    if (i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}


//CPU
void matrixMulCPU(float* res,const float *matrixA,const float *matrixB,int colsA,int rowsA,int rowsB) {
    float sum = 0;
    for (int i = 0; i < rowsA; ++i) {
        for (int j = 0; j < rowsB; ++j) {
            sum = 0;
            for (int k = 0; k < colsA; ++k) {
                sum += (float)matrixA[i*colsA+k]*(float)matrixB[k*rowsB+ j];
            }
            res[i*rowsB+j] = (float)sum;
        }
    }
}

// GPU
// C(i,j) = sum{A(i, k)* B(k ,j)}
// each thread cal C(i, j)
__global__ void matrixMulGPUKernal0(float* matrixC,const float* matrixA,const float *matrixB,int colsA,int rowsB) {
    
    float sum = 0;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    for (int i = 0; i < colsA; ++i) {
        sum += matrixA[row*colsA + i] * matrixB[i*rowsB + col];
    }
    matrixC[row*rowsB + col] = sum;
}

// Csub(i,j) = sum{A(i,ksub+offsetA)*B(ksub+offsetB,j)}  0 <= ksub < blockSize
// C(i,j) = sum{Csub(i,j)}
// each thread cal each block
__global__ void matrixMulGPUKernal1(float* matrixC,const float* matrixA,const float *matrixB,int colsA,int rowsB) {
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int aBegin = colsA*(by*BLOCK_SIZE);//A(0,by)
    int aEnd = aBegin + colsA - 1;
    int aStep = BLOCK_SIZE;//offsetA

    int bBegin = BLOCK_SIZE*bx;//B(bx,0)
    int bStep = BLOCK_SIZE*rowsB;//offsetB
    
    float cSub = 0;
    for (int a = aBegin,b = bBegin; a <= aEnd; a += aStep,b += bStep) {
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        
        As[ty][tx] = matrixA[a + colsA*ty + tx];
        Bs[ty][tx] = matrixB[b + rowsB*ty + tx];

        __syncthreads();
        
        //i * j for each thread
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            cSub += As[ty][k]*Bs[k][tx];
        }
        __syncthreads();
    }

    int cIndex = (by*BLOCK_SIZE + ty)*rowsB + (bx*BLOCK_SIZE + tx);
    matrixC[cIndex] = cSub;
}


void copyFromCPUToGPU(const float *matrixA, float *d_a, int n) {
    hipMemcpy(d_a, matrixA, sizeof(float) * n, hipMemcpyHostToDevice);
}

void copyFromGPUToCPU(const float *d_c, float *matrixC, int n) {
    hipMemcpy(matrixC, d_c, sizeof(float) * n, hipMemcpyDeviceToHost);
}

void matrixMulGPU(float* matrixC,const float *matrixA,const float *matrixB,int colsA,int rowsA,int rowsB) {

    float *d_a, *d_b, *d_c;
    hipMalloc((void**) &d_a, sizeof(float) * colsA*rowsA);   
    hipMalloc((void**) &d_b, sizeof(float) * rowsB*colsA);  
    hipMalloc((void**) &d_c, sizeof(float) * rowsB*rowsA); 

    copyFromCPUToGPU(matrixA,d_a,colsA*rowsA);
    copyFromCPUToGPU(matrixB,d_b,rowsB*colsA);

    dim3 blocks(rowsB/BLOCK_SIZE, rowsA/BLOCK_SIZE);
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

    float time_elapsed = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    matrixMulGPUKernal0<<<blocks,threads>>>(d_c,d_a,d_b,colsA,rowsA);

    hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf(" - Running time: %f ms\n", time_elapsed);
    double gflop = (2.0 * (double)colsA * colsA * colsA) * 0.000001;
    printf(" - GFlop: %.5f GFlop/sec\n\n", gflop/time_elapsed);

    hipDeviceSynchronize();
    copyFromGPUToCPU(d_c,matrixC,rowsB*rowsA);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}

void randomInit(float* _data,int size) {
    for (int i = 0; i < size; ++i) {
        _data[i] = rand()/(float)RAND_MAX;
    }
}

bool checkError(const float* matrixA, const float* matrixB, int size) {
    for (int i = 0 ; i < size; ++i) {
        if (fabs(matrixA[i] - matrixB[i]) > 1.0e-3) {
            printf(" ! Wrong index: %d\n", i);
            printf("%f \t %f\n",matrixA[i],matrixB[i]);
            return false;
        }
    }
    return true;
}

int main(int argc, char* argv[]) {

    if (!InitCUDA()) return 0; 
    srand(63);
    printf("\n - BLOCK_SIZE: %d\n", BLOCK_SIZE);

    int N = (1 << 11);
    int colsA, colsB, colsC, rowsA, rowsB, rowsC;
    colsA = colsB = colsC = rowsA = rowsB = rowsC = N;

    printf(" - Matrix size: %d * %d\n", rowsC, rowsC);

    float* A , *B, *C, *C2;
    A = (float*) malloc(sizeof(float) * colsA * rowsA);
    B = (float*) malloc(sizeof(float) * colsB * rowsB);

    randomInit(A,colsA*rowsA);
    randomInit(B,colsB*rowsB);

    C = (float*) malloc(sizeof(float) * colsC * rowsC);
    memset(C,0,sizeof(float)*colsC*rowsC);
    
    C2 = (float*) malloc(sizeof(float) * colsC * rowsC);
    memset(C2,0,sizeof(float)*colsC*rowsC);
    
    clock_t tick1 = clock();
    matrixMulCPU(C2,A,B,colsA,rowsA,colsB);
    printf(" - CPU use Time : %f ms\n",(double)(clock() - tick1)/CLOCKS_PER_SEC);

    // unsigned int timer = 0;
    // cutilCheckError(cutCreateTimer(&timer));
    // cutilCheckError(cutStartTimer(timer));
    matrixMulGPU(C,A,B,colsA,rowsA,colsB);
    // cutilCheckError(cutStopTimer(timer));
    // printf("GPU use time: %f (ms) \n", cutGetTimerValue(timer));
    // cutilCheckError(cutDeleteTimer(timer));

    if (checkError(C,C2,colsC*rowsC)) {
        printf("Right Answer!\n");
    }else{
        printf("Worng Answer!\n");
    }

    free(A);
    free(B);
    free(C);
    free(C2);

    return 0;
}
